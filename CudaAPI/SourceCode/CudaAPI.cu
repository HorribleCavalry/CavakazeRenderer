#include "hip/hip_runtime.h"
﻿#include "Common/Cuda3DMath.cuh"


//To solve the problem that can not use "CHECK" from another file in __global__ function, just choose the project setting->CUDA C/C++->Generate Relocatable Device Code.
//Refercenced website: https://www.cnblogs.com/qpswwww/p/11646593.html

__global__ void kernel()
{
	//CUM::vec2i vi0;
	//CUM::vec2i vi1;
	//CUM::vec2f vf0;
	//CUM::vec2f vf1;
	//vi0 = vi1;
	//vf0 = vi1;
	//vf1.x = 1.5;
	//vf0 = vf1;
	//vi0 = vf0;
	auto mat = CUM::Mat4x4_identity;
	CUM::Mat4x4i mati(5);
	mati += mat;
	CUM::Color3f color;
	CUM::calculateGammaColor(color, 2.2);
}


//class Person
//{
//public:
//	Person() {}
//	Person(const Person&) {}
//	Person(Person&&) {}
//	const Person& operator=(const Person&) {}
//	const Person& operator=(Person&&) {}
//	~Person() {}
//};

//__duel__ CUM::vec4<Float>&& reR()
//{
//	return CUM::vec4<Float>();
//}

int main()
{
	CUM::vec2i vi0;
	CUM::vec2i vi1(1.0f,2.0f);
	CUM::vec2f vf0;
	CUM::vec2f vf1(1.0f,2.0f);

	Int ni = 4;
	Float nf = 4.0;
	auto tempvi0 = vi0 + ni;
	auto tempvi1 = ni + vi0;
	auto tempvi2 = vi0 + nf;
	auto tempvi3 = nf + vi0;
	auto tempvi4 = vi0 + vi1;
	auto tempvi5 = vi0 + vf1;
	auto mat = CUM::Mat4x4_identity;
	CUM::Mat4x4i mati(5);
	mati += mat;
	mati -= mat;
	mati / mat;
	CUM::Mat4x4f matf;
	matf /= mat;
}