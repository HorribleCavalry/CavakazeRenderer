#include "hip/hip_runtime.h"
﻿#include "Common/Cuda3DMath.cuh"
#include "Common/CudaPrimitivesVector.cuh"
#include "Common/Tools.cuh"
#include "Common/Geometry/Geometry.cuh"
#include "cuda/std/limits"
#include <chrono>

//To solve the problem that can not use "CHECK" from another file in __global__ function, just choose the project setting->CUDA C/C++->Generate Relocatable Device Code.
//Refercenced website: https://www.cnblogs.com/qpswwww/p/11646593.html

class Person
{
public:
	__duel__ virtual void callType()
	{
		custd::OStream os;
		os << "I'm a person!" << custd::endl;
	}
};

class Student : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os<<"I'm a student!" << custd::endl;
	}
};

class Teacher : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a teacher!" << custd::endl;
	}
};

class Farmer : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Farmer!" << custd::endl;
	}
};

class Heacker : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Heacker!" << custd::endl;
	}
};

class Worker : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Worker!" << custd::endl;
	}
};

__global__ void kernel()
{
	CUM::PrimitiveVector<Geometry> geoVec;
	Sphere sp;
	OBox bx;
	geoVec.push_back(sp);
	geoVec.push_back(bx);
	for (Int i = 0; i < geoVec.Size(); i++)
	{
		geoVec[i].GetArea();
	}
}

//__duel__ CUM::Vec4<Float>&& reR()
//{
//	return CUM::Vec4<Float>();
//}

class Base
{
public:
	Float a;
	__duel__ virtual void Call()
	{
		printf("Called Base::Call()\n");
	}
	Float GetStaticVariable()
	{
		static Float val = a / 2.0;
		return val;
	}
};

class Child : public Base
{
public:
	Int b;
	__duel__ virtual void Call() override
	{
		printf("Called Child::Call()\n");
	}

};


template<typename T>
__global__ void testCopiedInstance(T* ins)
{
	custd::OStream os;
	os << ins->sampleTime<<custd::endl;
}

template<typename T>
__global__ void testSceneCopy(T* ins)
{
	ins->Call();
	ins->camera->Call();
	auto vecPtr = ins->primitivesVectorPtr;
	auto vec = *vecPtr;
	Int size = vec.Size();
	for (Int i = 0; i < size; i++)
	{
		vec[i].Call();
	}
}

int main()
{

	PersCamera persCam;
	CUM::PrimitiveVector<Geometry> vec;
	Geometry geo;
	Sphere sp;
	BBox bb;
	OBox ob;
	Triangle tri;
	vec.push_back(geo);
	vec.push_back(sp);
	vec.push_back(bb);
	vec.push_back(ob);
	vec.push_back(tri);
	Scene scene(&persCam, &vec);
	Scene* sceneDevice = scene.copyToDevice();
	testSceneCopy << <1, 1 >> > (sceneDevice);

	Base b0;
	b0.a = 1.0;
	Base b1;
	b1.a = 2.0;
	custd::cout << b0.GetStaticVariable() << custd::endl;
	custd::cout << b1.GetStaticVariable() << custd::endl;
	Int width = 5;
	Int height = 5;

	CUM::Color3f* buffer = new CUM::Color3f[width*height];
	Int idx = height / 2 * width + height / 2;
	buffer[idx] = CUM::Color3f(1.0);
	Texture texture(CUM::Vec2i(width, height), buffer);
	CUM::Color3f result = texture.GetColor(CUM::Vec2f(0.5, 0.5));
	custd::cout << result.r << custd::endl;
	delete[] buffer;
}