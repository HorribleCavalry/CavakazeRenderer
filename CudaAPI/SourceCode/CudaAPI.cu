#include "hip/hip_runtime.h"
﻿#include "Common/Cuda3DMath.cuh"
#include "Common/CudaPrimitivesVector.cuh"

//To solve the problem that can not use "CHECK" from another file in __global__ function, just choose the project setting->CUDA C/C++->Generate Relocatable Device Code.
//Refercenced website: https://www.cnblogs.com/qpswwww/p/11646593.html



class Person
{
public:
	__duel__ virtual void callType()
	{
		custd::OStream os;
		os << "I'm a person!" << custd::endl;
	}
};

class Student : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os<<"I'm a student!" << custd::endl;
	}
};

class Teacher : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a teacher!" << custd::endl;
	}
};

class Farmer : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Farmer!" << custd::endl;
	}
};

class Heacker : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Heacker!" << custd::endl;
	}
};

class Worker : public Person
{
public:
	__duel__ virtual void callType() override
	{
		custd::OStream os;
		os << "I'm a Worker!" << custd::endl;
	}
};

__global__ void kernel()
{
	Person per0;
	Student stu;
	Farmer far;
	Heacker hea;
	Worker wor;

	CUM::PrimitiveVector<Person> list;
	list.push_back(per0);
	list.push_back(stu);
	list.push_back(far);
	list.push_back(hea);
	list.push_back(wor);
	for (Int i = 0; i < 5; i++)
	{
		list[i].callType();
	}

	CUM::vec4f vf4(2.0f);
	auto test = CUM::normalize(vf4);
	CUM::vec3f vf41(2.0f);
	auto test1 = CUM::normalize(vf41);
}

//__duel__ CUM::vec4<Float>&& reR()
//{
//	return CUM::vec4<Float>();
//}

int main()
{
	//Person** prList = new Person*[5];

	//Person per0;
	//Student stu;
	//Farmer far;
	//Heacker hea;
	//Worker wor;
	//prList[0] = &per0;
	//prList[1] = &stu;
	//prList[2] = &far;
	//prList[3] = &hea;
	//prList[4] = &wor;
	//
	//for (int i = 0; i < 5; i++)
	//{
	//	prList[i]->callType();
	//}

	//CUM::PrimitiveVector<Person> list;
	//list.push_back(per0);
	//list.push_back(stu);
	//list.push_back(far);
	//list.push_back(hea);
	//list.push_back(wor);
	//for (Int i = 0; i < 5; i++)
	//{
	//	list[i].callType();
	//}

	kernel << <1, 1 >> > ();

	CUM::vec2i vi0;
	CUM::vec2i vi1(1.0f, 2.0f);
	CUM::vec2f vf0;
	CUM::vec2f vf1(1.0f, 2.0f);
	CUM::vec4f vf4(2.0f);
	auto test =CUM::normalize(vf4);
	CUM::vec3f vf41(2.0f);
	auto test1 = CUM::normalize(vf41);

	Person per0;
	Student stu;
	Farmer far;
	Heacker hea;
	Worker wor;

	CUM::PrimitiveVector<Person> list;
	list.push_back(per0);
	list.push_back(stu);
	list.push_back(far);
	list.push_back(hea);
	list.push_back(wor);
	for (Int i = 0; i < 5; i++)
	{
		list[i].callType();
	}

	//Int ni = 4;
	//Float nf = 4.0;
	//auto tempvi0 = vi0 + ni;
	//auto tempvi1 = ni + vi0;
	//auto tempvi2 = vi0 + nf;
	//auto tempvi3 = nf + vi0;
	//auto tempvi4 = vi0 + vi1;
	//auto tempvi5 = vi0 + vf1;
	//auto mat = CUM::Mat4x4_identity;
	//CUM::Mat4x4i mati(5);
	//mati += mat;
	//mati -= mat;
	//mati / mat;
	//CUM::Mat4x4f matf;
	//matf /= mat;
}